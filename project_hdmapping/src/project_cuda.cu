#include "hip/hip_runtime.h"
#include "project_cuda.h"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>

__global__ void identify_pixel_ranges(uint64_t* pixel_index, uint2* ranges,
                                        size_t *point_size, uint64_t threshold) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= *point_size)
		return;
    uint32_t pixel = pixel_index[idx] >> 32;
    if (pixel > threshold) {
        return;
    }
	if (idx == 0) {
		ranges[pixel].x = 0;
    }
	else {
		uint32_t pre_pixel = pixel_index[idx - 1] >> 32;
		if (pixel != pre_pixel) {
			ranges[pre_pixel].y = idx;
			ranges[pixel].x = idx;
		}
	}
	if (idx == *point_size - 1) {
		ranges[pixel].y = *point_size;
    }
}

__global__ void init_ranges(uint2* ranges, int *width, int *height) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= *width * *height) {
        return;
    }
    ranges[idx].x = 0;
    ranges[idx].y = 0;
}

__global__ void project_cuda(float *rotation, float *translation,
                             int *width, int *height, float *K,
                             float *points, size_t * point_size,
                             uint64_t *pixel_index, int *points_index, float *points_depth) {
    // TODO: pixel_inde points_index points_depth need reset
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= *point_size) {
        return;
    }
    // 可见idx是线性增加的，每隔四个存储空间是一个点的x, y, z, i
    // float3 是 CUDA 提供的一种 内置向量类型，用于存储 三维浮点数（x, y, z）
    float3 p_m = { points[4 * idx], points[4 * idx + 1], points[4 * idx + 2] };
    // to calculate p_c
    float3 p_c = {0, 0, 0};
    p_c.x = rotation[0] * p_m.x + rotation[1] * p_m.y + rotation[2] * p_m.z + translation[0];
    p_c.y = rotation[3] * p_m.x + rotation[4] * p_m.y + rotation[5] * p_m.z + translation[1];
    p_c.z = rotation[6] * p_m.x + rotation[7] * p_m.y + rotation[8] * p_m.z + translation[2];
    
    // 对相机系下的三维点进行过滤，不处理深度120米开外的点，和到相机光心距离130米之外的点
    if (p_c.z > 120 || p_c.z < 0 || (p_c.x * p_c.x + p_c.y * p_c.y + p_c.z + p_c.z) > 130 * 130) {
        return;
    }
    float3 p_cuni = {p_c.x / p_c.z, p_c.y / p_c.z, 1.0};
    float u = K[0] * p_cuni.x + K[1] * p_cuni.y + K[2];
    float v = K[3] * p_cuni.x + K[4] * p_cuni.y + K[5];
    
    // 强制类型转换将浮点型u, v转换成整型x, y
    int x = static_cast<int>(u);
    int y = static_cast<int>(v);

    // 对像素平面的像素坐标进行过滤，不处理在图像画幅之外的点
    if (x < 0 || x >= *width || y < 0 || y >= *height) {
        return;
    }

    // 得到x, y像素点的一维索引
    uint64_t pixel_index_i = static_cast<uint64_t>(y * *width + x);

    // 将pixel_index_i左移32位，低32位用来存储其他信息
    pixel_index_i <<= 32;

    // 低 32 位存储 p_c.z（当前点的深度值)
    pixel_index_i |= *((uint32_t*)&(p_c.z));

    // ************重要***********
    // 将像素(x, y)的一维索引存储到 pixel_index[idx(可以看做世界坐标系三维点的id)]
    pixel_index[idx] = pixel_index_i;

    // 将有效的idx(可以看做世界坐标系三维点的id)存储到points_index
    points_index[idx] = idx;
    
    // 将有效的idx(可以看做世界坐标系三维点的id)的深度值存储到points_depth
    points_depth[idx] = p_c.z;
}

__global__ void render(float* __restrict__ depth_image, unsigned char* __restrict__ intensity_image,
                       int* __restrict__ width, int* __restrict__ height,
                       float* __restrict__ points, uint2* __restrict__ ranges,
                       int* __restrict__ points_index, float* __restrict__ points_depth) {
    int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (pixel_x >= *width || pixel_y >= *height) {
        return;
    }
    int pixel_index = pixel_x + pixel_y * *width;
    uint2 range = ranges[pixel_index];
    if (range.y > range.x) {
        // float min_depth = 120.0;
        // float best_intensity = 0;
        // for (int k = range.x; k < range.y; k++) {
        //      // sum += *translation - bev_points[3 * points_index[k] + 2];
        //     float depth = points_depth[points_index[k]];
        //     if (depth < min_depth) {
        //         min_depth = depth;
        //         best_intensity = points[4 * points_index[k] + 3];
        //     }
        // }
        // depth_image[pixel_index] = min_depth;
        // intensity_image[pixel_index] = static_cast<unsigned char>(best_intensity);
        depth_image[pixel_index] = points_depth[points_index[range.x]];
        intensity_image[pixel_index] = static_cast<unsigned char>(points[4 * points_index[range.x] + 3]);
    }
    else {
        depth_image[pixel_index] = 0.0;
        intensity_image[pixel_index] = 0;
    }
}

ProjectorCUDA::ProjectorCUDA(float *points_w, size_t *point_size) : size_(*point_size) {
    
    // 给类的成员函数分配Device(GPU)内存

    hipMalloc((void**)&points_w_, *point_size * sizeof(float) * 4);
    hipMalloc((void**)&point_size_, sizeof(size_t));
    hipMalloc((void**)&pixel_index, size_ * sizeof(uint64_t));
    hipMalloc((void**)&points_index, size_ * sizeof(int));
    hipMalloc((void**)&pixel_index_sorted, size_ * sizeof(uint64_t));
    hipMalloc((void**)&points_index_sorted, size_ * sizeof(int));
    hipMalloc((void**)&points_depth, size_ * sizeof(float));

    // 将数据复制到GPU内存(显存)中,其中points_w_是成员变量，在显存中所占的存储空间为*point_size * sizeof(float) * 4

    hipMemcpy(points_w_, points_w, *point_size * sizeof(float) * 4, hipMemcpyHostToDevice);
    hipMemcpy(point_size_, point_size, sizeof(size_t), hipMemcpyHostToDevice);
}


void ProjectorCUDA::project(float *h_rotation, float *h_translation,
             int *h_width, int *h_height, float *h_K,
             float *h_depth_image, unsigned char *h_intensity_image) {
    // rasterization
    // pose and intrinsic
    // 初始化显存中的变量
    hipMemset(points_index, 0, size_ * sizeof(int));
    hipMemset(points_index_sorted, 0, size_ * sizeof(int));
    hipMemset(points_depth, 0, size_ * sizeof(float));

    // pixel_index为什么初始化成w * h + 1, size是所有3D点的数量？
    hipMemset(pixel_index, (*h_width * *h_height) + 1, size_ * sizeof(uint64_t));
    hipMemset(pixel_index_sorted, 0, size_ * sizeof(uint64_t));
    uint2* ranges;
    float *rotation, *translation;
    int *width, *height;
    float *K;
    float *depth_image;
    unsigned char *intensity_image;

    hipMalloc((void**)&ranges, (*h_width) * (*h_width) * sizeof(uint2));

    hipMalloc((void**)&rotation, 9 * sizeof(float));
    hipMalloc((void**)&translation, 3 * sizeof(float));
    hipMalloc((void**)&width, sizeof(int));
    hipMalloc((void**)&height, sizeof(int));
    hipMalloc((void**)&K, 9 * sizeof(float));
    hipMalloc((void**)&depth_image, (*h_width) * (*h_height) * sizeof(float));
    hipMalloc((void**)&intensity_image, (*h_width) * (*h_height) * sizeof(unsigned char));

    hipMemcpy(rotation, h_rotation, 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(translation, h_translation, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(width, h_width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(height, h_width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(K, h_K, 9 * sizeof(float), hipMemcpyHostToDevice);

    project_cuda<<< (size_ + 255) / 256, 256 >>>(rotation, translation,
                                                 width, height, K,
                                                 points_w_, point_size_,
                                                 pixel_index, points_index, points_depth);

    hipDeviceSynchronize();
    void *sort_storage_tmp = nullptr;
    size_t sort_storage_size = 0;
    hipcub::DeviceRadixSort::SortPairs(sort_storage_tmp, sort_storage_size, pixel_index, pixel_index_sorted, points_index, points_index_sorted, size_);
    hipMalloc(&sort_storage_tmp, sort_storage_size);
    hipcub::DeviceRadixSort::SortPairs(sort_storage_tmp, sort_storage_size, pixel_index, pixel_index_sorted, points_index, points_index_sorted, size_);
    hipDeviceSynchronize();
    // test<<< 1, 1 >>>(pixel_index_sorted, points_index_sorted);
    hipFree(sort_storage_tmp);
    init_ranges<<< (*h_width * *h_height + 255) / 256 , 256>>>(ranges, width, height);
    identify_pixel_ranges<<< (size_ + 255) / 256, 256 >>> (pixel_index_sorted, ranges, point_size_, *h_width * *h_height);
    dim3 block_dim(16, 16);
    dim3 grid_dim((*h_width + block_dim.x - 1) / block_dim.x, (*h_height + block_dim.y - 1) / block_dim.y);

    render<<< grid_dim, block_dim >>>(depth_image, intensity_image,
                              width, height, points_w_, ranges, points_index_sorted, points_depth);

    hipDeviceSynchronize();

    hipMemcpy(h_depth_image, depth_image, (*h_height * *h_width) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_intensity_image, intensity_image, (*h_height * *h_width) * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // hipFree(pixel_index);
    // hipFree(points_index);
    // hipFree(pixel_index_sorted);
    // hipFree(points_index_sorted);
    // hipFree(points_depth);
    hipFree(ranges);
    hipFree(rotation);
    hipFree(translation);
    hipFree(width);
    hipFree(height);
    hipFree(K);
    hipFree(depth_image);
    hipFree(intensity_image);
    hipMemset(depth_image, 0, (*h_height * *h_width) * sizeof(float));
    hipMemset(intensity_image, 0, (*h_height * *h_width) * sizeof(unsigned char));
}

void ProjectorCUDA::filter_pts(float *projectionMatching, bool *hasPointProjected, 
                               bool *isPointVisible, int *L) {
    // 先弄清楚需要哪些成员变量，可以在这里进行赋值
    hipMemset();
    float 

}

ProjectorCUDA::~ProjectorCUDA() {
    hipFree(points_w_);
    hipFree(point_size_);
    hipFree(pixel_index);
    hipFree(points_index);
    hipFree(pixel_index_sorted);
    hipFree(points_index_sorted);
    hipFree(points_depth);
}

__global__ void ComputePointVisibility_cuda(bool *isPointVisible, 
                                           float *projectionMatching,
                                           bool *hasPointProjected, 
                                           int *width, int *height, int *L)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= *point_size) {
        return;
    }

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= H * W) return;  // If the thread index exceeds the total number of pixels, return.

    int i = idx / W;  // Row index
    int j = idx % W;  // Column index

    if (hasPointProjected[idx])
    {
        Eigen::Vector2d Xpx(i, j);
        Eigen::Vector3d P = projectionMatching[idx];

        int umin = max(0, i - L); int vmin = max(0, j - L);
        int umax = min(H - 1, i + L); int vmax = min(W - 1, j + L);

        double solidAnglesPerSector[8];
        bool isSectorEmpty[8];
        
        // Initialize the solidAnglesPerSector with large values and mark all sectors as empty.
        for (int k = 0; k < 8; ++k)
        {
            solidAnglesPerSector[k] = std::numeric_limits<double>::max();
            isSectorEmpty[k] = true;
        }

        for (int u = umin; u <= umax; ++u)
        {
            for (int v = vmin; v <= vmax; ++v)
            {
                if (u == i && v == j) continue;

                int neighborIdx = u * W + v;
                if (hasPointProjected[neighborIdx])
                {
                    Eigen::Vector2d Ypx(u, v) - Xpx;
                    double polar2DAngle = std::atan2(Ypx(1), Ypx(0)) + 3.14159265359;
                    int sectorIdx = floor(4.0 * polar2DAngle / 3.14159265359);

                    Eigen::Vector3d Q = projectionMatching[neighborIdx];
                    Eigen::Vector3d PQ = Q - P;
                    Eigen::Vector3d PO = -1.0 * P;
                    double angle = std::abs(SignedAngle(PO, PQ));

                    if (angle < solidAnglesPerSector[sectorIdx])
                    {
                        solidAnglesPerSector[sectorIdx] = angle;
                        isSectorEmpty[sectorIdx] = false;
                    }
                }
            }
        }

        double sumAngle = 0;
        unsigned int nbrNonEmptySector = 0;
        unsigned int nbrSectorNonVisible = 0;
        for (int sectorIdx = 0; sectorIdx < 8; ++sectorIdx)
        {
            if (!isSectorEmpty[sectorIdx])
            {
                sumAngle += solidAnglesPerSector[sectorIdx];
                nbrNonEmptySector++;
                if (solidAnglesPerSector[sectorIdx] < 0.25)
                {
                    nbrSectorNonVisible++;
                }
            }
        }

        if (sumAngle < 2.0)
        {
            isPointVisible[idx] = false;
        }
    }
}

void LaunchComputePointVisibilityCUDA(bool* isPointVisible,
                                      const Eigen::Vector3d* projectionMatching,
                                      const bool* hasPointProjected, 
                                      int H, int W, int L)
{
    int blockSize = 256;  // Number of threads per block
    int numBlocks = (H * W + blockSize - 1) / blockSize;  // Number of blocks needed

    // Launch the kernel to compute visibility
    ComputePointVisibilityCUDA<<<numBlocks, blockSize>>>(isPointVisible, projectionMatching, hasPointProjected, H, W, L);

    // Wait for kernel to finish
    hipDeviceSynchronize();
}
